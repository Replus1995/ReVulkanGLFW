#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ValkanCudaApp.h"


// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return ((unsigned int)(rgba.w * 255.0f) << 24) |
         ((unsigned int)(rgba.z * 255.0f) << 16) |
         ((unsigned int)(rgba.y * 255.0f) << 8) |
         ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c) {
  float4 rgba;
  rgba.x = (c & 0xff) * 0.003921568627f;          //  /255.0f;
  rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;   //  /255.0f;
  rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;  //  /255.0f;
  rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;  //  /255.0f;
  return rgba;
}

// row pass using texture lookups
__global__ void d_boxfilter_rgba_x(hipSurfaceObject_t* dstSurfMipMapArray,
                                   hipTextureObject_t textureMipMapInput,
                                   size_t baseWidth, size_t baseHeight,
                                   size_t mipLevels, int filter_radius) {
  float scale = 1.0f / (float)((filter_radius << 1) + 1);
  unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y < baseHeight) {
    for (uint32_t mipLevelIdx = 0; mipLevelIdx < mipLevels; mipLevelIdx++) {
      uint32_t width =
          (baseWidth >> mipLevelIdx) ? (baseWidth >> mipLevelIdx) : 1;
      uint32_t height =
          (baseHeight >> mipLevelIdx) ? (baseHeight >> mipLevelIdx) : 1;
      if (y < height && filter_radius < width) {
        float px = 1.0 / width;
        float py = 1.0 / height;
        float4 t = make_float4(0.0f);
        for (int x = -filter_radius; x <= filter_radius; x++) {
          t += tex2DLod<float4>(textureMipMapInput, x * px, y * py,
                                (float)mipLevelIdx);
        }

        unsigned int dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], 0, y);

        for (int x = 1; x < width; x++) {
          t += tex2DLod<float4>(textureMipMapInput, (x + filter_radius) * px,
                                y * py, (float)mipLevelIdx);
          t -=
              tex2DLod<float4>(textureMipMapInput, (x - filter_radius - 1) * px,
                               y * py, (float)mipLevelIdx);
          unsigned int dataB = rgbaFloatToInt(t * scale);
          surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx],
                      x * sizeof(uchar4), y);
        }
      }
    }
  }
}

// column pass using coalesced global memory reads
__global__ void d_boxfilter_rgba_y(hipSurfaceObject_t* dstSurfMipMapArray,
                                   hipSurfaceObject_t* srcSurfMipMapArray,
                                   size_t baseWidth, size_t baseHeight,
                                   size_t mipLevels, int filter_radius) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  float scale = 1.0f / (float)((filter_radius << 1) + 1);

  for (uint32_t mipLevelIdx = 0; mipLevelIdx < mipLevels; mipLevelIdx++) {
    uint32_t width =
        (baseWidth >> mipLevelIdx) ? (baseWidth >> mipLevelIdx) : 1;
    uint32_t height =
        (baseHeight >> mipLevelIdx) ? (baseHeight >> mipLevelIdx) : 1;

    if (x < width && height > filter_radius) {
      float4 t;
      // do left edge
      int colInBytes = x * sizeof(uchar4);
      unsigned int pixFirst = surf2Dread<unsigned int>(
          srcSurfMipMapArray[mipLevelIdx], colInBytes, 0);
      t = rgbaIntToFloat(pixFirst) * filter_radius;

      for (int y = 0; (y < (filter_radius + 1)) && (y < height); y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y);
        t += rgbaIntToFloat(pix);
      }

      unsigned int dataB = rgbaFloatToInt(t * scale);
      surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, 0);

      for (int y = 1; (y < filter_radius + 1) && ((y + filter_radius) < height);
           y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y + filter_radius);
        t += rgbaIntToFloat(pix);
        t -= rgbaIntToFloat(pixFirst);

        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }

      // main loop
      for (int y = (filter_radius + 1); y < (height - filter_radius); y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y + filter_radius);
        t += rgbaIntToFloat(pix);

        pix = surf2Dread<unsigned int>(srcSurfMipMapArray[mipLevelIdx],
                                       colInBytes, y - filter_radius - 1);
        t -= rgbaIntToFloat(pix);

        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }

      // do right edge
      unsigned int pixLast = surf2Dread<unsigned int>(
          srcSurfMipMapArray[mipLevelIdx], colInBytes, height - 1);
      for (int y = height - filter_radius;
           (y < height) && ((y - filter_radius - 1) > 1); y++) {
        t += rgbaIntToFloat(pixLast);
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y - filter_radius - 1);
        t -= rgbaIntToFloat(pix);
        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }
    }
  }
}




void vulkanImageCUDA::cudaUpdateVkImage()
{
	cudaVkSemaphoreWait(cudaExtVkUpdateCudaSemaphore);

	int nthreads = 128;

	/*Perform 2D box filter on image using CUDA */
	d_boxfilter_rgba_x << <m_imageHeight / nthreads, nthreads, 0, streamToRun >> > (
		d_surfaceObjectListTemp, textureObjMipMapInput, m_imageWidth, m_imageHeight,
		mipLevels, filter_radius);

	d_boxfilter_rgba_y << <m_imageWidth / nthreads, nthreads, 0, streamToRun >> > (
		d_surfaceObjectList, d_surfaceObjectListTemp, m_imageWidth, m_imageHeight,
		mipLevels, filter_radius);

	varySigma();

	cudaVkSemaphoreSignal(cudaExtCudaUpdateVkSemaphore);
}
